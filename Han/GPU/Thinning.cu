#include "hip/hip_runtime.h"
// Thinning.cu
// 实现二值图像的细化算法

#include "Thinning.h"
#include <iostream>
#include <stdio.h>
using namespace std;

// 宏：DEF_BLOCK_X 和 DEF_BLOCK_Y
// 定义了默认的线程块尺寸。
#define DEF_BLOCK_X  32
#define DEF_BLOCK_Y   8

#define uchar unsigned char

// 宏：DEF_PATTERN_SIZE
// 定义了 PATTERN 表的默认大小。
#define DEF_PATTERN_SIZE  512

#define HIGH 255
#define LOW 0
#define MoreThanOne(x1, x2, x3, x4, x5, x6, x7, x8, qt) ( (x1) >= qt || (x2) >= qt || (x3) >= qt || (x4) >= qt || (x5) >= qt || (x6) >= qt || (x7) >= qt || (x8) >= qt ? 1 : 0)

#define notZero2(x1, x2) ( (x1) != 0 && (x2) != 0 ? 1 : 0)
#define notZero3(x1, x2, x3) ( (x1) != 0 && (x2) != 0 && (x3) != 0 ? 1 : 0)
#define notZero4(x1, x2, x3, x4) ( (x1) != 0 && (x2) != 0 && (x3) != 0 && (x4) != 0 ? 1 : 0)
#define notZero5(x1, x2, x3, x4, x5) ( (x1) != 0 && (x2) != 0 && (x3) != 0 && (x4) != 0 && (x5) != 0 ? 1 : 0)
#define notZero6(x1, x2, x3, x4, x5, x6) ( (x1) != 0 && (x2) != 0 && (x3) != 0 && (x4) != 0 && (x5) != 0 && (x6) != 0 ? 1 : 0)
#define notZero7(x1, x2, x3, x4, x5, x6, x7) ( (x1) != 0 && (x2) != 0 && (x3) != 0 && (x4) != 0 && (x5) != 0 && (x6) != 0 && (x7) != 0 ? 1 : 0)

static __global__ void _calWightKer(ImageCuda inimg, ImageCuda weight)
{
    // c 和 r 分别表示线程处理的像素点的坐标的 x 和 y 分量 （其中，c 表示
    // column，r 表示 row ）。
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，一方面节省计算资源，
    // 另一方面防止由于段错误导致程序崩溃。
    // 两边各有两个点不处理。
    if (c >= inimg.imgMeta.width - 1 || 
         r >= inimg.imgMeta.height - 1 || c < 1 || r < 1)
        return;

    // 定义目标点位置的指针。
    unsigned char *outptr;

    // 获取当前像素点在图像中的相对位置。
    // 从左上角第二行第二列开始计算。
    int curpos = (r) * inimg.pitchBytes + c ;

    // 获取当前像素点在图像中的绝对位置。
    outptr = inimg.imgMeta.imgData + curpos ;

    if (*outptr == HIGH) {
        unsigned char x1 = inimg.imgMeta.imgData[curpos - inimg.pitchBytes - 1];
        unsigned char x2 = inimg.imgMeta.imgData[curpos - inimg.pitchBytes];
        unsigned char x3 = inimg.imgMeta.imgData[curpos - inimg.pitchBytes + 1];
        unsigned char x4 = inimg.imgMeta.imgData[curpos - 1];
        unsigned char x5 = inimg.imgMeta.imgData[curpos + 1];
        unsigned char x6 = inimg.imgMeta.imgData[curpos + inimg.pitchBytes - 1];
        unsigned char x7 = inimg.imgMeta.imgData[curpos + inimg.pitchBytes];
        unsigned char x8 = inimg.imgMeta.imgData[curpos + inimg.pitchBytes + 1];

        weight.imgMeta.imgData[curpos] = (x1 == HIGH) + (x2 == HIGH) + (x3 == HIGH) + (x4 == HIGH) + 
                                         (x5 == HIGH) + (x6 == HIGH) + (x7 == HIGH) + (x8 == HIGH);
    }
}

static __global__ void _thinHanKer(ImageCuda weight, ImageCuda outimg, int *devchangecount)
{
    // c 和 r 分别表示线程处理的像素点的坐标的 x 和 y 分量 （其中，c 表示
    // column，r 表示 row ）。
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，一方面节省计算资源，
    // 另一方面防止由于段错误导致程序崩溃。
    // 两边各有两个点不处理。
    if (c >= outimg.imgMeta.width - 1 || 
         r >= outimg.imgMeta.height - 1 || c < 1 || r < 1)
        return;

    // 定义目标点位置的指针。
    unsigned char *outptr;

    // 获取当前像素点在图像中的相对位置。
    // 从左上角第二行第二列开始计算。
    int curpos = (r) * outimg.pitchBytes + c ;

    // 获取当前像素点在图像中的绝对位置。
    outptr = outimg.imgMeta.imgData + curpos ;
    

    // 忽略低像素点情况和dot情况
    if (*outptr == HIGH && (weight.imgMeta.imgData[curpos] > 0 && weight.imgMeta.imgData[curpos] < 8)) {
        uchar x1 = weight.imgMeta.imgData[curpos - outimg.pitchBytes - 1];
        uchar x2 = weight.imgMeta.imgData[curpos - outimg.pitchBytes];
        uchar x3 = weight.imgMeta.imgData[curpos - outimg.pitchBytes + 1];
        uchar x4 = weight.imgMeta.imgData[curpos + 1];
        uchar x5 = weight.imgMeta.imgData[curpos + outimg.pitchBytes + 1];
        uchar x6 = weight.imgMeta.imgData[curpos + outimg.pitchBytes];
        uchar x7 = weight.imgMeta.imgData[curpos + outimg.pitchBytes - 1];
        uchar x8 = weight.imgMeta.imgData[curpos - 1];

        // 判断每一个点的权重值
        switch (weight.imgMeta.imgData[curpos])
        {
        case 1 :
            if (MoreThanOne(x1,x2,x3,x4,x5,x6,x7,x8,3))
            {
                *outptr = LOW;
                *devchangecount = 1;
            }
            break;
        case 2 : 
            if (MoreThanOne(x1,x2,x3,x4,x5,x6,x7,x8,3))
                if (x1!=0&&x2!=0 || x2!=0&&x3!=0 || x3!=0&&x4!=0 || x4!=0&&x5!=0 || x5!=0&&x6!=0 || x6!=0&&x7!=0 ||
                    x7!=0&&x8!=0 || x8!=0&&x1!=0 || x2!=0&&x4!=0 || x4!=0&&x6!=0 || x6!=0&&x8!=0 || x8!=0&&x2!=0)
                {
                    *outptr = LOW;
                    *devchangecount = 1;
                }
            break;
        case 3 : 
            if (MoreThanOne(x1,x2,x3,x4,x5,x6,x7,x8,7))
                if (notZero3(x6,x7,x8) || notZero3(x1,x2,x3) || notZero3(x1,x7,x8) || notZero3(x6,x7,x5) || 
                    notZero3(x3,x4,x5) || notZero3(x2,x3,x4) || notZero3(x4,x5,x6) || notZero3(x8,x1,x2) ||
                    notZero3(x6,x7,x4) || notZero3(x6,x1,x8) || notZero3(x6,x3,x4) || notZero3(x6,x5,x8))
                {
                    *outptr = LOW;
                    *devchangecount = 1;
                }
            break;
        case 4 : 
            if (notZero4(x1,x2,x3,x4) || notZero4(x1,x2,x7,x8) || notZero4(x1,x2,x3,x8) || notZero4(x1,x6,x7,x8) ||
                notZero4(x5,x6,x7,x8) || notZero4(x4,x5,x6,x7) || notZero4(x3,x4,x5,x6) || notZero4(x5,x2,x3,x4) ||
                notZero4(x6,x7,x3,x4) || notZero4(x1,x8,x5,x6))
            {
                *outptr = LOW;
                    *devchangecount = 1;
            }
            break;
        case 5 : 
            if (x1==8 || x2==8 || x3==8 || x4==8 || x5==8 || x6==8 || x7==8 || x8==8)
                if (notZero5(x7,x8,x1,x2,x3) || notZero5(x7,x8,x1,x5,x6) || notZero5(x3,x4,x5,x6,x7) || notZero5(x1,x2,x3,x4,x5) ||
                    notZero5(x4,x5,x6,x7,x8) || notZero5(x6,x7,x8,x1,x2) || notZero5(x1,x2,x3,x4,x8) || notZero5(x2,x3,x4,x5,x6)) {
                    *outptr = LOW;
                    *devchangecount = 1;
                }
            break;
        case 6 : 
            if (x1==8 || x2==8 || x3==8 || x4==8 || x5==8 || x6==8 || x7==8 || x8==8)
                if (notZero6(x3,x4,x5,x6,x7,x8) || notZero6(x1,x2,x3,x6,x7,x8) || notZero6(x1,x2,x5,x6,x7,x8) || notZero6(x1,x4,x5,x6,x7,x8) ||
                    notZero6(x1,x2,x3,x4,x7,x8) || notZero6(x3,x4,x5,x6,x7,x2) || notZero6(x3,x4,x5,x6,x1,x2) || notZero6(x1,x2,x3,x4,x5,x8)) {
                    *outptr = LOW;
                    *devchangecount = 1;
                }
            break;
        case 7 : 
            if (x1==8 || x2==8 || x3==8 || x4==8 || x5==8 || x6==8 || x7==8 || x8==8)
                if (notZero7(x1,x2,x3,x5,x6,x7,x8) || notZero7(x1,x3,x4,x5,x6,x7,x8) ||
                    notZero7(x1,x2,x3,x4,x5,x6,x7) || notZero7(x1,x2,x3,x4,x5,x7,x8)){
                    *outptr = LOW;
                    *devchangecount = 1;
                }
            break;
        default:
            break;
        }
    }
}

// 直接并行化
// 线程数，处理多少个点有多少线程数
__host__ int Thinning::thinHan(Image *inimg, Image *outimg)
{
    // 局部变量，错误码。
    int errcode;  
    hipError_t cudaerrcode; 

    // 检查输入图像，输出图像是否为空。
    if (inimg == NULL || outimg == NULL)
        return NULL_POINTER;

    // 声明所有中间变量并初始化为空。
    Image *weight = NULL;
    int *devchangecount = NULL;

    // 记录细化点数的变量，位于 host 端。
    int changeCount;

    // 记录细化点数的变量，位于 device 端。并为其申请空间。
    cudaerrcode = hipMalloc((void **)&devchangecount, sizeof (int));
    if (cudaerrcode != hipSuccess) {
        // FAIL_THIN_IMAGE_FREE;
        return CUDA_ERROR;
    }

    // 生成暂存图像。
    errcode = ImageBasicOp::newImage(&weight);
    if (errcode != NO_ERROR)
        return errcode;
    errcode = ImageBasicOp::makeAtCurrentDevice(weight, inimg->width, 
                                                inimg->height);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 将输入图像 inimg 完全拷贝到输出图像 outimg ，并将 outimg 拷贝到 
    // device 端。
    errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 提取输出图像
    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 提取暂存图像
    ImageCuda weightsubimgCud;
    errcode = ImageBasicOp::roiSubImage(weight, &weightsubimgCud);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 计算调用 Kernel 函数的线程块的尺寸和线程块的数量。
    dim3 gridsize, blocksize;
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y - 1) / blocksize.y;

    /*gridsize.x = 1;*/
    /*gridsize.y = 1;//(outsubimgCud.imgMeta.height + blocksize.y - 1) / blocksize.y;*/
    // 赋值为 1，以便开始第一次迭代。
    changeCount = 1;

    // 开始迭代，当不可再被细化，即记录细化点数的变量 changeCount 的值为 0 时，
    // 停止迭代。 
    while (changeCount > 0) {
        // 将 host 端的变量赋值为 0 ，并将值拷贝到 device 端的 devchangecount。
        changeCount = 0;
        cudaerrcode = hipMemcpy(devchangecount, &changeCount, sizeof (int),
                                 hipMemcpyHostToDevice);
        if (cudaerrcode != hipSuccess) {
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }

        // 初始化 weight 为 0
        cudaerrcode = hipMemset (weight->imgData, 0, sizeof(unsigned char) * weight->width * weight->height);
        if (cudaerrcode != hipSuccess) {
         return CUDA_ERROR;
        }
        
        _calWightKer<<<gridsize, blocksize>>>(outsubimgCud, weightsubimgCud);
        if (hipGetLastError() != hipSuccess) {
            // 核函数出错，结束迭代函数，释放申请的变量空间。
            return CUDA_ERROR;
        }

        // 调用核函数，开始第一步细化操作。
        _thinHanKer<<<gridsize, blocksize>>>(weightsubimgCud, outsubimgCud, devchangecount);
        if (hipGetLastError() != hipSuccess) {
            // 核函数出错，结束迭代函数，释放申请的变量空间。
            return CUDA_ERROR;
        }    
        
        // 将位于 device 端的 devchangecount 拷贝到 host 端上的 changeCount 
        // 变量，进行迭代判断。
        cudaerrcode = hipMemcpy(&changeCount, devchangecount, sizeof (int),
                                 hipMemcpyDeviceToHost);
        if (cudaerrcode != hipSuccess) {
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }

   }

    // 细化结束后释放申请的变量空间。
    hipFree(devchangecount);
    ImageBasicOp::deleteImage(weight);

    return NO_ERROR;
}

// GPU版本2，优化分支，使用Pattern表法，Pattern表位于global内存中
static __global__ void _thinHanPtKer(ImageCuda weight, ImageCuda outimg, 
                                    int *devchangecount, unsigned char *dev_lut)
{
    // c 和 r 分别表示线程处理的像素点的坐标的 x 和 y 分量 （其中，c 表示
    // column，r 表示 row ）。
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，一方面节省计算资源，
    // 另一方面防止由于段错误导致程序崩溃。
    // 两边各有两个点不处理。
    if (c >= outimg.imgMeta.width - 1 || 
         r >= outimg.imgMeta.height - 1 || c < 1 || r < 1)
        return;

    // 定义目标点位置的指针。
    unsigned char *outptr;

    // 获取当前像素点在图像中的相对位置。
    // 从左上角第二行第二列开始计算。
    int curpos = (r) * outimg.pitchBytes + c ;

    // 获取当前像素点在图像中的绝对位置。
    outptr = outimg.imgMeta.imgData + curpos ;
    

    // 忽略低像素点情况和dot情况
    if (*outptr == HIGH && (weight.imgMeta.imgData[curpos] > 0 && weight.imgMeta.imgData[curpos] < 8)) {
        uchar x1 = weight.imgMeta.imgData[curpos - outimg.pitchBytes - 1];
        uchar x2 = weight.imgMeta.imgData[curpos - outimg.pitchBytes];
        uchar x3 = weight.imgMeta.imgData[curpos - outimg.pitchBytes + 1];
        uchar x4 = weight.imgMeta.imgData[curpos + 1];
        uchar x5 = weight.imgMeta.imgData[curpos + outimg.pitchBytes + 1];
        uchar x6 = weight.imgMeta.imgData[curpos + outimg.pitchBytes];
        uchar x7 = weight.imgMeta.imgData[curpos + outimg.pitchBytes - 1];
        uchar x8 = weight.imgMeta.imgData[curpos - 1];
        uchar index = (x1!=0) * 1 + (x2!=0) * 2 + (x3!=0) * 4 + (x4!=0) * 8 + 
                      (x5!=0) * 16 + (x6!=0) * 32 + (x7!=0) * 64 + (x8!=0) * 128; 

        // 判断每一个点的权重值
        switch (weight.imgMeta.imgData[curpos])
        {
        case 1 :
            if (MoreThanOne(x1,x2,x3,x4,x5,x6,x7,x8,3))
            {
                *outptr = LOW;
                *devchangecount = 1;
            }
            break;
        case 2 : 
            if (MoreThanOne(x1,x2,x3,x4,x5,x6,x7,x8,3))
                if (dev_lut[index])
                {
                    *outptr = LOW;
                    *devchangecount = 1;
                }
            break;
        case 3 : 
            if (MoreThanOne(x1,x2,x3,x4,x5,x6,x7,x8,7))
                if (dev_lut[index + 256])
                {
                    *outptr = LOW;
                    *devchangecount = 1;
                }
            break;
        case 4 : 
            if (dev_lut[index + 512])
            {
                *outptr = LOW;
                    *devchangecount = 1;
            }
            break;
        case 5 : 
            if (x1==8 || x2==8 || x3==8 || x4==8 || x5==8 || x6==8 || x7==8 || x8==8)
                if (dev_lut[index + 768]) {
                    *outptr = LOW;
                    *devchangecount = 1;
                }
            break;
        case 6 : 
            if (x1==8 || x2==8 || x3==8 || x4==8 || x5==8 || x6==8 || x7==8 || x8==8)
                if (dev_lut[index + 1024]) {
                    *outptr = LOW;
                    *devchangecount = 1;
                }
            break;
        case 7 : 
            if (x1==8 || x2==8 || x3==8 || x4==8 || x5==8 || x6==8 || x7==8 || x8==8)
                if (dev_lut[index + 1280]){
                    *outptr = LOW;
                    *devchangecount = 1;
                }
            break;
        default:
            break;
        }
    }
}

__host__ int Thinning::thinHanPt(Image *inimg, Image *outimg)
{
    // 局部变量，错误码。
    int errcode;  
    hipError_t cudaerrcode; 

    // 检查输入图像，输出图像是否为空。
    if (inimg == NULL || outimg == NULL)
        return NULL_POINTER;

    unsigned char *dev_lut;
    unsigned char lut[1536] = 
    {
        0, 0, 0, 1, 0, 0, 1, 1, 0, 0, 1, 1, 1, 1, 1, 1, 0, 0, 0, 1, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 0, 1, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
        0, 0, 0, 1, 0, 0, 1, 1, 0, 0, 1, 1, 1, 1, 1, 1, 0, 0, 0, 1, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 1, 1, 1, 0, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 0, 1, 1, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 

        0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 1, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 1, 1, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 1, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 1, 1, 1, 1, 
        0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 
        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,

        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 
        0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 1, 1, 
        0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 

        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 
        0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 

        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 
        0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 1, 0, 1, 1, 1, 1, 1, 

        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 
        0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1
    };
    cudaerrcode = hipMalloc((void **)&dev_lut, sizeof (unsigned char) * 1536);
    if (cudaerrcode != hipSuccess) 
        return CUDA_ERROR;

    cudaerrcode = hipMemcpy(dev_lut, lut, sizeof(unsigned char) * 1536, hipMemcpyHostToDevice);
    if (cudaerrcode != hipSuccess) 
        return CUDA_ERROR;


    // 声明所有中间变量并初始化为空。
    Image *weight = NULL;
    int *devchangecount = NULL;

    // 记录细化点数的变量，位于 host 端。
    int changeCount;

    // 记录细化点数的变量，位于 device 端。并为其申请空间。
    cudaerrcode = hipMalloc((void **)&devchangecount, sizeof (int));
    if (cudaerrcode != hipSuccess) {
        // FAIL_THIN_IMAGE_FREE;
        return CUDA_ERROR;
    }

    // 生成暂存图像。
    errcode = ImageBasicOp::newImage(&weight);
    if (errcode != NO_ERROR)
        return errcode;
    errcode = ImageBasicOp::makeAtCurrentDevice(weight, inimg->width, 
                                                inimg->height);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 将输入图像 inimg 完全拷贝到输出图像 outimg ，并将 outimg 拷贝到 
    // device 端。
    errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 提取输出图像
    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 提取暂存图像
    ImageCuda weightsubimgCud;
    errcode = ImageBasicOp::roiSubImage(weight, &weightsubimgCud);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 计算调用 Kernel 函数的线程块的尺寸和线程块的数量。
    dim3 gridsize, blocksize;
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y - 1) / blocksize.y;

    // 赋值为 1，以便开始第一次迭代。
    changeCount = 1;

    // 开始迭代，当不可再被细化，即记录细化点数的变量 changeCount 的值为 0 时，
    // 停止迭代。 
    while (changeCount > 0) {
        // 将 host 端的变量赋值为 0 ，并将值拷贝到 device 端的 devchangecount。
        changeCount = 0;
        cudaerrcode = hipMemcpy(devchangecount, &changeCount, sizeof (int),
                                 hipMemcpyHostToDevice);
        if (cudaerrcode != hipSuccess) {
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }

        // 初始化 weight 为 0
        cudaerrcode = hipMemset (weight->imgData, 0, sizeof(unsigned char) * weight->width * weight->height);
        if (cudaerrcode != hipSuccess) {
         return CUDA_ERROR;
        }
        
        _calWightKer<<<gridsize, blocksize>>>(outsubimgCud, weightsubimgCud);
        if (hipGetLastError() != hipSuccess) {
            // 核函数出错，结束迭代函数，释放申请的变量空间。
            return CUDA_ERROR;
        }

        // 调用核函数，开始第一步细化操作。
        _thinHanPtKer<<<gridsize, blocksize>>>(weightsubimgCud, outsubimgCud, devchangecount, dev_lut);
        if (hipGetLastError() != hipSuccess) {
            // 核函数出错，结束迭代函数，释放申请的变量空间。
            return CUDA_ERROR;
        }    
        
        // 将位于 device 端的 devchangecount 拷贝到 host 端上的 changeCount 
        // 变量，进行迭代判断。
        cudaerrcode = hipMemcpy(&changeCount, devchangecount, sizeof (int),
                                 hipMemcpyDeviceToHost);
        if (cudaerrcode != hipSuccess) {
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }

   }

    // 细化结束后释放申请的变量空间。
    hipFree(devchangecount);
    ImageBasicOp::deleteImage(weight);

    return NO_ERROR;
}
/*
// GPU版本3，优化读取，将Pattern存到常量内存中
__constant__ unsigned char con_lutthin[256] = { 0, 0, 0, 1, 0, 0, 1, 1, 0, 1, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 1, 0, 1, 1, 
    0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 
    0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
    0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 1, 1, 
    0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
    0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 1, 1, 
    0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 0, 0, 0, 1, 1, 0, 0, 0, 1, 0, 1, 1, 
    1, 1, 0, 0, 1, 1, 0, 0 };

static __global__ void _thinHanPtConKer(ImageCuda tempimg, ImageCuda outimg, int *devchangecount)
{
     // c 和 r 分别表示线程处理的像素点的坐标的 x 和 y 分量 （其中，c 表示
    // column，r 表示 row ）。
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，一方面节省计算资源，
    // 另一方面防止由于段错误导致程序崩溃。
    // 两边各有两个点不处理。
    if (c >= outimg.imgMeta.width - 2 || 
         r >= outimg.imgMeta.height - 2 || c < 2 || r < 2)
        return;

    // 定义目标点位置的指针。
    unsigned char *outptr;

    // 获取当前像素点在图像中的相对位置。
    // 从左上角第二行第二列开始计算。
    int curpos = (r) * outimg.pitchBytes + c ;

    // 获取当前像素点在图像中的绝对位置。
    outptr = outimg.imgMeta.imgData + curpos ;
    unsigned char x1, x2, x3, x4, x5, x6, x7, x8;

    // 如果目标像素点的像素值为低像素, 则不进行细化处理。
    if (isHigh1(*outptr)) {
        x1 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes - 1];
        x2 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes];
        x3 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes + 1];
        x4 = tempimg.imgMeta.imgData[curpos - 1];
        x5 = tempimg.imgMeta.imgData[curpos + 1];
        x6 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes - 1];
        x7 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes];
        x8 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes + 1];
        unsigned char x9,x10,x11;
        if(isHigh1(x7)) {
            x9 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2 - 1];
            x10 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2];
            x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2 + 1];

            if (isHigh5(x4,x5,x6,x7,x8) && isLow2(x2,x10) ||
                isHigh3(x6,x7,x9) && isLow8(x1,x2,x3,x4,x5,x8,x10,x11) ||
                isHigh3(x7,x8,x11) && isLow8(x1,x2,x3,x4,x5,x6,x9,x10))
                    return ;
        } 
        if(isHigh1(x2)) {
            // w is down
            x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2 - 1];
            x10 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2];
            x11 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2 + 1];

            if (isHigh5(x1,x2,x3,x4,x5) && isLow2(x7,x10)){
                outimg.imgMeta.imgData[curpos] = LOW;
                *devchangecount = 1;
                return ;
            } else if (isHigh3(x1,x2,x9) && isLow8(x3,x4,x5,x6,x7,x8,x10,x11) ||
                       isHigh3(x2,x3,x11) && isLow8(x1,x4,x5,x6,x7,x8,x9,x10)) 
                return ;
        }
        if(isHigh1(x5)) {
            x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes + 2];
            x10 = tempimg.imgMeta.imgData[curpos + 2];
            x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes + 2];

            if (isHigh5(x2,x3,x5,x7,x8) && isLow2(x4,x10) ||
                isHigh3(x5,x8,x11) && isLow8(x1,x2,x3,x4,x6,x7,x9,x10) ||
                isHigh3(x3,x5,x9) && isLow8(x1,x2,x4,x6,x7,x8,x10,x11))
                return ;
        }
        if(isHigh1(x4)){
            x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes - 2];
            x10 = tempimg.imgMeta.imgData[curpos - 2];
            x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes - 2];
            if (isHigh5(x1,x2,x4,x6,x7) && isLow2(x5,x10)){
                outimg.imgMeta.imgData[curpos] = LOW;
                *devchangecount = 1;
                return ;
            } else if (isHigh3(x4,x6,x11) && isLow8(x1,x2,x3,x5,x7,x8,x9,x10) ||
                       isHigh3(x1,x4,x9) && isLow8(x2,x3,x5,x6,x7,x8,x10,x11)) 
                return ;
        }

        // 1   2   4
        // 8       16
        // 32  64  128
        unsigned char index = isHigh1(x1) * 1 + isHigh1(x2) * 2 + isHigh1(x3) * 4 + isHigh1(x4) * 8 +
                              isHigh1(x5) * 16 + isHigh1(x6) * 32 + isHigh1(x7) * 64 + isHigh1(x8) * 128;
        
        if (con_lutthin[index] == 1) {
                outimg.imgMeta.imgData[curpos] = LOW;
                *devchangecount = 1;
        }       
    }
}

__host__ int Thinning::thinHanPtCon(Image *inimg, Image *outimg)
{
 // 局部变量，错误码。
    int errcode;  
    hipError_t cudaerrcode; 

    // 检查输入图像，输出图像是否为空。
    if (inimg == NULL || outimg == NULL)
        return NULL_POINTER;

    // 声明所有中间变量并初始化为空。
    Image *tempimg = NULL;
    int *devchangecount = NULL;

    // 记录细化点数的变量，位于 host 端。
    int changeCount;

    // 记录细化点数的变量，位于 device 端。并为其申请空间。
    cudaerrcode = hipMalloc((void **)&devchangecount, sizeof (int));
    if (cudaerrcode != hipSuccess) {
        // FAIL_THIN_IMAGE_FREE;
        return CUDA_ERROR;
    }

    // 生成暂存图像。
    errcode = ImageBasicOp::newImage(&tempimg);
    if (errcode != NO_ERROR)
        return errcode;
    errcode = ImageBasicOp::makeAtCurrentDevice(tempimg, inimg->width, 
                                                inimg->height);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 将输入图像 inimg 完全拷贝到输出图像 outimg ，并将 outimg 拷贝到 
    // device 端。
    errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 提取输出图像
    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 提取暂存图像
    ImageCuda tempsubimgCud;
    errcode = ImageBasicOp::roiSubImage(tempimg, &tempsubimgCud);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 计算调用 Kernel 函数的线程块的尺寸和线程块的数量。
    dim3 gridsize, blocksize;
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y - 1) / blocksize.y;

    // 赋值为 1，以便开始第一次迭代。
    changeCount = 1;

    // 开始迭代，当不可再被细化，即记录细化点数的变量 changeCount 的值为 0 时，
    // 停止迭代。 
    while (changeCount > 0) {
        // 将 host 端的变量赋值为 0 ，并将值拷贝到 device 端的 devchangecount。
        changeCount = 0;
        cudaerrcode = hipMemcpy(devchangecount, &changeCount, sizeof (int),
                                 hipMemcpyHostToDevice);
        if (cudaerrcode != hipSuccess) {
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }

        // copy ouimg to tempimg 
         cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
                                      outimg->imgData, outsubimgCud.deviceId, 
                                      outsubimgCud.pitchBytes * outimg->height);
        
         if (cudaerrcode != hipSuccess) {
             return CUDA_ERROR;
         }
            
        // 调用核函数，开始第一步细化操作。
        _thinHanPtConKer<<<gridsize, blocksize>>>(tempsubimgCud, outsubimgCud, devchangecount);
        if (hipGetLastError() != hipSuccess) {
            // 核函数出错，结束迭代函数，释放申请的变量空间。
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }    
        
        // 将位于 device 端的 devchangecount 拷贝到 host 端上的 changeCount 
        // 变量，进行迭代判断。
        cudaerrcode = hipMemcpy(&changeCount, devchangecount, sizeof (int),
                                 hipMemcpyDeviceToHost);
        if (cudaerrcode != hipSuccess) {
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }

   }

    // 细化结束后释放申请的变量空间。
    hipFree(devchangecount);
    ImageBasicOp::deleteImage(tempimg);

    return NO_ERROR;
}

// GPU版本5，GPU版本1的基础上使用单线程处理四个点
static __global__ void _thinHanFourKer(ImageCuda tempimg, ImageCuda outimg, int *devchangecount)
{
    // c 和 r 分别表示线程处理的像素点的坐标的 x 和 y 分量 （其中，c 表示
    // column，r 表示 row ）。
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，一方面节省计算资源，
    // 另一方面防止由于段错误导致程序崩溃。
    // 两边各有两个点不处理。
    if (c >= outimg.imgMeta.width - 2 || 
         r >= outimg.imgMeta.height - 2 || c < 2 || r < 2)
        return;

    // 定义目标点位置的指针。
    unsigned char *outptr;

    // 获取当前像素点在图像中的相对位置。
    // 从左上角第二行第二列开始计算。
    int curpos = (r) * outimg.pitchBytes + c ;

    // 获取当前像素点在图像中的绝对位置。
    outptr = outimg.imgMeta.imgData + curpos ;
    unsigned char x1, x2, x3, x4, x5, x6, x7, x8;

    // 如果目标像素点的像素值为低像素, 则不进行细化处理。
    if (isHigh1(*outptr)) {
        x1 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes - 1];
        x2 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes];
        x3 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes + 1];
        x4 = tempimg.imgMeta.imgData[curpos - 1];
        x5 = tempimg.imgMeta.imgData[curpos + 1];
        x6 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes - 1];
        x7 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes];
        x8 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes + 1];
        unsigned char x9,x10,x11;
        if(isHigh1(x7)) {
            x9 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2 - 1];
            x10 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2];
            x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2 + 1];

            if (isHigh5(x4,x5,x6,x7,x8) && isLow2(x2,x10) ||
                isHigh3(x6,x7,x9) && isLow8(x1,x2,x3,x4,x5,x8,x10,x11) ||
                isHigh3(x7,x8,x11) && isLow8(x1,x2,x3,x4,x5,x6,x9,x10))
                    return ;
        } 
        if(isHigh1(x2)) {
            // w is down
            x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2 - 1];
            x10 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2];
            x11 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2 + 1];

            if (isHigh5(x1,x2,x3,x4,x5) && isLow2(x7,x10)){
                outimg.imgMeta.imgData[curpos] = LOW;
                *devchangecount = 1;
                return ;
            } else if (isHigh3(x1,x2,x9) && isLow8(x3,x4,x5,x6,x7,x8,x10,x11) ||
                       isHigh3(x2,x3,x11) && isLow8(x1,x4,x5,x6,x7,x8,x9,x10)) 
                return ;
        }
        if(isHigh1(x5)) {
            x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes + 2];
            x10 = tempimg.imgMeta.imgData[curpos + 2];
            x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes + 2];

            if (isHigh5(x2,x3,x5,x7,x8) && isLow2(x4,x10) ||
                isHigh3(x5,x8,x11) && isLow8(x1,x2,x3,x4,x6,x7,x9,x10) ||
                isHigh3(x3,x5,x9) && isLow8(x1,x2,x4,x6,x7,x8,x10,x11))
                return ;
        }
        if(isHigh1(x4)){
            x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes - 2];
            x10 = tempimg.imgMeta.imgData[curpos - 2];
            x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes - 2];
            if (isHigh5(x1,x2,x4,x6,x7) && isLow2(x5,x10)){
                outimg.imgMeta.imgData[curpos] = LOW;
                *devchangecount = 1;
                return ;
            } else if (isHigh3(x4,x6,x11) && isLow8(x1,x2,x3,x5,x7,x8,x9,x10) ||
                       isHigh3(x1,x4,x9) && isLow8(x2,x3,x5,x6,x7,x8,x10,x11)) 
                return ;
        }
        
        if (isHigh4(x1, x4, x6, x7) && isLow2(x3, x5) || // 1
            isHigh4(x1, x2, x4, x6) && isLow2(x5, x8) || // 2
            isHigh4(x1, x2, x3, x5) && isLow2(x6, x7) || // 3
            isHigh4(x1, x2, x3, x4) && isLow2(x7, x8) || // 4
            isLow4(x3, x5, x7, x8) && isHigh2(x1, x4) || // 5
            isLow4(x5, x6, x7, x8) && isHigh2(x1, x2) || // 6
            isLow1(x5) && isHigh7(x1, x2, x3, x4, x6, x7, x8) || // 7
            isLow1(x7) && isHigh7(x1, x2, x3, x4, x5, x6, x8) || // 8
            isLow4(x2, x3, x5, x8) && isHigh2(x4, x6) || // 9
            isLow4(x1, x2, x3, x5) && isHigh2(x6, x7) || // 10
            isLow4(x4, x6, x7, x8) && isHigh2(x2, x3) || // 11
            isLow4(x1, x4, x6, x7) && isHigh2(x3, x5) || // 12
            isLow4(x1, x2, x3, x4) && isHigh2(x7, x8) || // 13
            isLow4(x1, x2, x4, x6) && isHigh2(x5, x8) || // 14
            isLow1(x4) && isHigh7(x1, x2, x3, x5, x6, x7, x8) || // 15
            isLow1(x2) && isHigh7(x1, x3, x4, x5, x6, x7, x8) || // 16
            isHigh4(x3, x5, x7, x8) && isLow2(x1, x4) || // 17
            isHigh4(x2, x3, x5, x8) && isLow2(x4, x6) || // 18
            isHigh4(x5, x6, x7, x8) && isLow2(x1, x2) || // 19
            isHigh4(x4, x6, x7, x8) && isLow2(x2, x3) // 20
            ) {
                outimg.imgMeta.imgData[curpos] = LOW;
                *devchangecount = 1;
        }       
    }

    for (int i = 0; i < 3; ++i) {
        if (++r >= outimg.imgMeta.height - 2)
            return ;
        curpos += outimg.pitchBytes;  

        // 获取当前像素点在图像中的绝对位置。
        outptr = outimg.imgMeta.imgData + curpos;

        // 如果目标像素点的像素值为低像素, 则不进行细化处理。
        if (isHigh1(*outptr)) {
            x1 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes - 1];
            x2 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes];
            x3 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes + 1];
            x4 = tempimg.imgMeta.imgData[curpos - 1];
            x5 = tempimg.imgMeta.imgData[curpos + 1];
            x6 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes - 1];
            x7 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes];
            x8 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes + 1];
            unsigned char x9,x10,x11;
            if(isHigh1(x7)) {
                x9 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2 - 1];
                x10 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2];
                x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2 + 1];

                if (isHigh5(x4,x5,x6,x7,x8) && isLow2(x2,x10) ||
                    isHigh3(x6,x7,x9) && isLow8(x1,x2,x3,x4,x5,x8,x10,x11) ||
                    isHigh3(x7,x8,x11) && isLow8(x1,x2,x3,x4,x5,x6,x9,x10))
                        return ;
            } 
            if(isHigh1(x2)) {
                // w is down
                x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2 - 1];
                x10 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2];
                x11 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2 + 1];

                if (isHigh5(x1,x2,x3,x4,x5) && isLow2(x7,x10)){
                    outimg.imgMeta.imgData[curpos] = LOW;
                    *devchangecount = 1;
                    return ;
                } else if (isHigh3(x1,x2,x9) && isLow8(x3,x4,x5,x6,x7,x8,x10,x11) ||
                           isHigh3(x2,x3,x11) && isLow8(x1,x4,x5,x6,x7,x8,x9,x10)) 
                    return ;
            }
            if(isHigh1(x5)) {
                x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes + 2];
                x10 = tempimg.imgMeta.imgData[curpos + 2];
                x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes + 2];

                if (isHigh5(x2,x3,x5,x7,x8) && isLow2(x4,x10) ||
                    isHigh3(x5,x8,x11) && isLow8(x1,x2,x3,x4,x6,x7,x9,x10) ||
                    isHigh3(x3,x5,x9) && isLow8(x1,x2,x4,x6,x7,x8,x10,x11))
                    return ;
            }
            if(isHigh1(x4)){
                x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes - 2];
                x10 = tempimg.imgMeta.imgData[curpos - 2];
                x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes - 2];
                if (isHigh5(x1,x2,x4,x6,x7) && isLow2(x5,x10)){
                    outimg.imgMeta.imgData[curpos] = LOW;
                    *devchangecount = 1;
                    return ;
                } else if (isHigh3(x4,x6,x11) && isLow8(x1,x2,x3,x5,x7,x8,x9,x10) ||
                           isHigh3(x1,x4,x9) && isLow8(x2,x3,x5,x6,x7,x8,x10,x11)) 
                    return ;
            }
            
            if (isHigh4(x1, x4, x6, x7) && isLow2(x3, x5) || // 1
                isHigh4(x1, x2, x4, x6) && isLow2(x5, x8) || // 2
                isHigh4(x1, x2, x3, x5) && isLow2(x6, x7) || // 3
                isHigh4(x1, x2, x3, x4) && isLow2(x7, x8) || // 4
                isLow4(x3, x5, x7, x8) && isHigh2(x1, x4) || // 5
                isLow4(x5, x6, x7, x8) && isHigh2(x1, x2) || // 6
                isLow1(x5) && isHigh7(x1, x2, x3, x4, x6, x7, x8) || // 7
                isLow1(x7) && isHigh7(x1, x2, x3, x4, x5, x6, x8) || // 8
                isLow4(x2, x3, x5, x8) && isHigh2(x4, x6) || // 9
                isLow4(x1, x2, x3, x5) && isHigh2(x6, x7) || // 10
                isLow4(x4, x6, x7, x8) && isHigh2(x2, x3) || // 11
                isLow4(x1, x4, x6, x7) && isHigh2(x3, x5) || // 12
                isLow4(x1, x2, x3, x4) && isHigh2(x7, x8) || // 13
                isLow4(x1, x2, x4, x6) && isHigh2(x5, x8) || // 14
                isLow1(x4) && isHigh7(x1, x2, x3, x5, x6, x7, x8) || // 15
                isLow1(x2) && isHigh7(x1, x3, x4, x5, x6, x7, x8) || // 16
                isHigh4(x3, x5, x7, x8) && isLow2(x1, x4) || // 17
                isHigh4(x2, x3, x5, x8) && isLow2(x4, x6) || // 18
                isHigh4(x5, x6, x7, x8) && isLow2(x1, x2) || // 19
                isHigh4(x4, x6, x7, x8) && isLow2(x2, x3) // 20
                ) {
                    outimg.imgMeta.imgData[curpos] = LOW;
                    *devchangecount = 1;
            }       
        }

    }
}

__host__ int Thinning::thinHanFour(Image *inimg, Image *outimg)
{
    // 局部变量，错误码。
    int errcode;  
    hipError_t cudaerrcode; 

    // 检查输入图像，输出图像是否为空。
    if (inimg == NULL || outimg == NULL)
        return NULL_POINTER;

    // 声明所有中间变量并初始化为空。
    Image *tempimg = NULL;
    int *devchangecount = NULL;

    // 记录细化点数的变量，位于 host 端。
    int changeCount;

    // 记录细化点数的变量，位于 device 端。并为其申请空间。
    cudaerrcode = hipMalloc((void **)&devchangecount, sizeof (int));
    if (cudaerrcode != hipSuccess) {
        // FAIL_THIN_IMAGE_FREE;
        return CUDA_ERROR;
    }

    // 生成暂存图像。
    errcode = ImageBasicOp::newImage(&tempimg);
    if (errcode != NO_ERROR)
        return errcode;
    errcode = ImageBasicOp::makeAtCurrentDevice(tempimg, inimg->width, 
                                                inimg->height);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 将输入图像 inimg 完全拷贝到输出图像 outimg ，并将 outimg 拷贝到 
    // device 端。
    errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 提取输出图像
    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 提取暂存图像
    ImageCuda tempsubimgCud;
    errcode = ImageBasicOp::roiSubImage(tempimg, &tempsubimgCud);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 计算调用 Kernel 函数的线程块的尺寸和线程块的数量。
    dim3 gridsize, blocksize;
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y * 4 - 1) / blocksize.y * 4;

    // 赋值为 1，以便开始第一次迭代。
    changeCount = 1;

    // 开始迭代，当不可再被细化，即记录细化点数的变量 changeCount 的值为 0 时，
    // 停止迭代。 
    while (changeCount > 0) {
        // 将 host 端的变量赋值为 0 ，并将值拷贝到 device 端的 devchangecount。
        changeCount = 0;
        cudaerrcode = hipMemcpy(devchangecount, &changeCount, sizeof (int),
                                 hipMemcpyHostToDevice);
        if (cudaerrcode != hipSuccess) {
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }

        // copy ouimg to tempimg 
         cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
                                      outimg->imgData, outsubimgCud.deviceId, 
                                      outsubimgCud.pitchBytes * outimg->height);
        
         if (cudaerrcode != hipSuccess) {
             return CUDA_ERROR;
         }
            
        // 调用核函数，开始第一步细化操作。
        _thinHanFourKer<<<gridsize, blocksize>>>(tempsubimgCud, outsubimgCud, devchangecount);
        if (hipGetLastError() != hipSuccess) {
            // 核函数出错，结束迭代函数，释放申请的变量空间。
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }    
        
        // 将位于 device 端的 devchangecount 拷贝到 host 端上的 changeCount 
        // 变量，进行迭代判断。
        cudaerrcode = hipMemcpy(&changeCount, devchangecount, sizeof (int),
                                 hipMemcpyDeviceToHost);
        if (cudaerrcode != hipSuccess) {
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }

   }

    // 细化结束后释放申请的变量空间。
    hipFree(devchangecount);
    ImageBasicOp::deleteImage(tempimg);

    return NO_ERROR;
}

// GPU版本4，使用Pattern表法后，单线程内逻辑大大降低。使单线程处理四个点，增大线程复杂度，减少线程数量
static __global__ void _thinHanPtConFourKer(ImageCuda tempimg, ImageCuda outimg, int *devchangecount)
{
     // c 和 r 分别表示线程处理的像素点的坐标的 x 和 y 分量 （其中，c 表示
    // column，r 表示 row ）。
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;

    // 检查第一个像素点是否越界，如果越界，则不进行处理，一方面节省计算资源，
    // 另一方面防止由于段错误导致程序崩溃。
    // 两边各有两个点不处理。
    if (c >= outimg.imgMeta.width - 2 || 
         r >= outimg.imgMeta.height - 2 || c < 2 || r < 2)
        return;

    // 定义目标点位置的指针。
    unsigned char *outptr;

    // 获取当前像素点在图像中的相对位置。
    // 从左上角第二行第二列开始计算。
    int curpos = (r) * outimg.pitchBytes + c ;

    // 获取当前像素点在图像中的绝对位置。
    outptr = outimg.imgMeta.imgData + curpos ;
    unsigned char x1, x2, x3, x4, x5, x6, x7, x8;

    // 如果目标像素点的像素值为低像素, 则不进行细化处理。
    if (isHigh1(*outptr)) {
        x1 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes - 1];
        x2 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes];
        x3 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes + 1];
        x4 = tempimg.imgMeta.imgData[curpos - 1];
        x5 = tempimg.imgMeta.imgData[curpos + 1];
        x6 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes - 1];
        x7 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes];
        x8 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes + 1];
        unsigned char x9,x10,x11;
        if(isHigh1(x7)) {
            x9 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2 - 1];
            x10 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2];
            x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2 + 1];

            if (isHigh5(x4,x5,x6,x7,x8) && isLow2(x2,x10) ||
                isHigh3(x6,x7,x9) && isLow8(x1,x2,x3,x4,x5,x8,x10,x11) ||
                isHigh3(x7,x8,x11) && isLow8(x1,x2,x3,x4,x5,x6,x9,x10))
                    return ;
        } 
        if(isHigh1(x2)) {
            // w is down
            x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2 - 1];
            x10 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2];
            x11 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2 + 1];

            if (isHigh5(x1,x2,x3,x4,x5) && isLow2(x7,x10)){
                outimg.imgMeta.imgData[curpos] = LOW;
                *devchangecount = 1;
                return ;
            } else if (isHigh3(x1,x2,x9) && isLow8(x3,x4,x5,x6,x7,x8,x10,x11) ||
                       isHigh3(x2,x3,x11) && isLow8(x1,x4,x5,x6,x7,x8,x9,x10)) 
                return ;
        }
        if(isHigh1(x5)) {
            x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes + 2];
            x10 = tempimg.imgMeta.imgData[curpos + 2];
            x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes + 2];

            if (isHigh5(x2,x3,x5,x7,x8) && isLow2(x4,x10) ||
                isHigh3(x5,x8,x11) && isLow8(x1,x2,x3,x4,x6,x7,x9,x10) ||
                isHigh3(x3,x5,x9) && isLow8(x1,x2,x4,x6,x7,x8,x10,x11))
                return ;
        }
        if(isHigh1(x4)){
            x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes - 2];
            x10 = tempimg.imgMeta.imgData[curpos - 2];
            x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes - 2];
            if (isHigh5(x1,x2,x4,x6,x7) && isLow2(x5,x10)){
                outimg.imgMeta.imgData[curpos] = LOW;
                *devchangecount = 1;
                return ;
            } else if (isHigh3(x4,x6,x11) && isLow8(x1,x2,x3,x5,x7,x8,x9,x10) ||
                       isHigh3(x1,x4,x9) && isLow8(x2,x3,x5,x6,x7,x8,x10,x11)) 
                return ;
        }

        // 1   2   4
        // 8       16
        // 32  64  128
        unsigned char index = isHigh1(x1) * 1 + isHigh1(x2) * 2 + isHigh1(x3) * 4 + isHigh1(x4) * 8 +
                              isHigh1(x5) * 16 + isHigh1(x6) * 32 + isHigh1(x7) * 64 + isHigh1(x8) * 128;
        
        if (con_lutthin[index] == 1) {
                outimg.imgMeta.imgData[curpos] = LOW;
                *devchangecount = 1;
        }       
    }

    for (int i = 0; i < 3; ++i) {
        if (++r >= outimg.imgMeta.height - 2)
            return ;
        curpos += outimg.pitchBytes;  

        // 获取当前像素点在图像中的绝对位置。
        outptr = outimg.imgMeta.imgData + curpos;

        // 如果目标像素点的像素值为低像素, 则不进行细化处理。
        if (isHigh1(*outptr)) {
            x1 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes - 1];
            x2 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes];
            x3 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes + 1];
            x4 = tempimg.imgMeta.imgData[curpos - 1];
            x5 = tempimg.imgMeta.imgData[curpos + 1];
            x6 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes - 1];
            x7 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes];
            x8 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes + 1];
            unsigned char x9,x10,x11;
            if(isHigh1(x7)) {
                x9 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2 - 1];
                x10 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2];
                x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes * 2 + 1];

                if (isHigh5(x4,x5,x6,x7,x8) && isLow2(x2,x10) ||
                    isHigh3(x6,x7,x9) && isLow8(x1,x2,x3,x4,x5,x8,x10,x11) ||
                    isHigh3(x7,x8,x11) && isLow8(x1,x2,x3,x4,x5,x6,x9,x10))
                        return ;
            } 
            if(isHigh1(x2)) {
                // w is down
                x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2 - 1];
                x10 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2];
                x11 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes * 2 + 1];

                if (isHigh5(x1,x2,x3,x4,x5) && isLow2(x7,x10)){
                    outimg.imgMeta.imgData[curpos] = LOW;
                    *devchangecount = 1;
                    return ;
                } else if (isHigh3(x1,x2,x9) && isLow8(x3,x4,x5,x6,x7,x8,x10,x11) ||
                           isHigh3(x2,x3,x11) && isLow8(x1,x4,x5,x6,x7,x8,x9,x10)) 
                    return ;
            }
            if(isHigh1(x5)) {
                x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes + 2];
                x10 = tempimg.imgMeta.imgData[curpos + 2];
                x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes + 2];

                if (isHigh5(x2,x3,x5,x7,x8) && isLow2(x4,x10) ||
                    isHigh3(x5,x8,x11) && isLow8(x1,x2,x3,x4,x6,x7,x9,x10) ||
                    isHigh3(x3,x5,x9) && isLow8(x1,x2,x4,x6,x7,x8,x10,x11))
                    return ;
            }
            if(isHigh1(x4)){
                x9 = tempimg.imgMeta.imgData[curpos - outimg.pitchBytes - 2];
                x10 = tempimg.imgMeta.imgData[curpos - 2];
                x11 = tempimg.imgMeta.imgData[curpos + outimg.pitchBytes - 2];
                if (isHigh5(x1,x2,x4,x6,x7) && isLow2(x5,x10)){
                    outimg.imgMeta.imgData[curpos] = LOW;
                    *devchangecount = 1;
                    return ;
                } else if (isHigh3(x4,x6,x11) && isLow8(x1,x2,x3,x5,x7,x8,x9,x10) ||
                           isHigh3(x1,x4,x9) && isLow8(x2,x3,x5,x6,x7,x8,x10,x11)) 
                    return ;
            }

            // 1   2   4
            // 8       16
            // 32  64  128
            unsigned char index = isHigh1(x1) * 1 + isHigh1(x2) * 2 + isHigh1(x3) * 4 + isHigh1(x4) * 8 +
                                  isHigh1(x5) * 16 + isHigh1(x6) * 32 + isHigh1(x7) * 64 + isHigh1(x8) * 128;
            
            if (con_lutthin[index] == 1) {
                    outimg.imgMeta.imgData[curpos] = LOW;
                    *devchangecount = 1;
            }       
        }
    }
}

__host__ int Thinning::thinHanPtConFour(Image *inimg, Image *outimg)
{
 // 局部变量，错误码。
    int errcode;  
    hipError_t cudaerrcode; 

    // 检查输入图像，输出图像是否为空。
    if (inimg == NULL || outimg == NULL)
        return NULL_POINTER;

    // 声明所有中间变量并初始化为空。
    Image *tempimg = NULL;
    int *devchangecount = NULL;

    // 记录细化点数的变量，位于 host 端。
    int changeCount;

    // 记录细化点数的变量，位于 device 端。并为其申请空间。
    cudaerrcode = hipMalloc((void **)&devchangecount, sizeof (int));
    if (cudaerrcode != hipSuccess) {
        // FAIL_THIN_IMAGE_FREE;
        return CUDA_ERROR;
    }

    // 生成暂存图像。
    errcode = ImageBasicOp::newImage(&tempimg);
    if (errcode != NO_ERROR)
        return errcode;
    errcode = ImageBasicOp::makeAtCurrentDevice(tempimg, inimg->width, 
                                                inimg->height);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 将输入图像 inimg 完全拷贝到输出图像 outimg ，并将 outimg 拷贝到 
    // device 端。
    errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 提取输出图像
    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 提取暂存图像
    ImageCuda tempsubimgCud;
    errcode = ImageBasicOp::roiSubImage(tempimg, &tempsubimgCud);
    if (errcode != NO_ERROR) {
        // FAIL_THIN_IMAGE_FREE;
        return errcode;
    }

    // 计算调用 Kernel 函数的线程块的尺寸和线程块的数量。
    dim3 gridsize, blocksize;
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y * 4 - 1) / blocksize.y * 4;

    // 赋值为 1，以便开始第一次迭代。
    changeCount = 1;

    // 开始迭代，当不可再被细化，即记录细化点数的变量 changeCount 的值为 0 时，
    // 停止迭代。 
    while (changeCount > 0) {
        // 将 host 端的变量赋值为 0 ，并将值拷贝到 device 端的 devchangecount。
        changeCount = 0;
        cudaerrcode = hipMemcpy(devchangecount, &changeCount, sizeof (int),
                                 hipMemcpyHostToDevice);
        if (cudaerrcode != hipSuccess) {
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }

        // copy ouimg to tempimg 
         cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
                                      outimg->imgData, outsubimgCud.deviceId, 
                                      outsubimgCud.pitchBytes * outimg->height);
        
         if (cudaerrcode != hipSuccess) {
             return CUDA_ERROR;
         }
            
        // 调用核函数，开始第一步细化操作。
        _thinHanPtConFourKer<<<gridsize, blocksize>>>(tempsubimgCud, outsubimgCud, devchangecount);
        if (hipGetLastError() != hipSuccess) {
            // 核函数出错，结束迭代函数，释放申请的变量空间。
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }    
        
        // 将位于 device 端的 devchangecount 拷贝到 host 端上的 changeCount 
        // 变量，进行迭代判断。
        cudaerrcode = hipMemcpy(&changeCount, devchangecount, sizeof (int),
                                 hipMemcpyDeviceToHost);
        if (cudaerrcode != hipSuccess) {
            // FAIL_THIN_IMAGE_FREE;
            return CUDA_ERROR;
        }

   }

    // 细化结束后释放申请的变量空间。
    hipFree(devchangecount);
    ImageBasicOp::deleteImage(tempimg);

    return NO_ERROR;
}
*/