#include <iostream>
#include "Thinning.h"
#include "ErrorCode.h"
#include "Image.h"
using namespace std;

int main(int argc, char const **argv)
{
	if(argc < 2)
	{
		cout << "Please input image!" << endl;
		return 0;
	}
	Thinning thin_gpu;


	Image *inimg;
    ImageBasicOp::newImage(&inimg);
    int errcode;
    errcode = ImageBasicOp::readFromFile(argv[1], inimg);
    if (errcode != NO_ERROR) {
        cout << "error: " << errcode << endl;
        return 0; 
    }
    for(int i = 0; i < inimg->width * inimg->height; i++)
    {
        if(inimg->imgData[i] != 0)
            inimg->imgData[i] = 255;
    }
    Image *outimg1;
    ImageBasicOp::newImage(&outimg1);
    ImageBasicOp::makeAtHost(outimg1, inimg->width, inimg->height);

    Image *outimg2;
    ImageBasicOp::newImage(&outimg2);
    ImageBasicOp::makeAtHost(outimg2, inimg->width, inimg->height);

    Image *outimg3;
    ImageBasicOp::newImage(&outimg3);
    ImageBasicOp::makeAtHost(outimg3, inimg->width, inimg->height);

    Image *outimg4;
    ImageBasicOp::newImage(&outimg4);
    ImageBasicOp::makeAtHost(outimg4, inimg->width, inimg->height);

    Image *outimg5;
    ImageBasicOp::newImage(&outimg5);
    ImageBasicOp::makeAtHost(outimg5, inimg->width, inimg->height);

    Image *outimg6;
    ImageBasicOp::newImage(&outimg6);
    ImageBasicOp::makeAtHost(outimg6, inimg->width, inimg->height);
    
    hipEvent_t start, stop;
    float runTime;

    // 直接并行
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < 100; i++) 
       thin_gpu.thinHan(inimg, outimg1);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runTime, start, stop);
	cout << "A2() time is " << (runTime) / 100 << " ms" << endl;
    ImageBasicOp::copyToHost(outimg1);
    ImageBasicOp::writeToFile("A2_outimg.bmp", outimg1); 

    // Pattern 表法，Pattern位于 global 内存
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // float runTime;
    hipEventRecord(start, 0);
    for (int i = 0; i < 100; i++) 
        thin_gpu.thinHanPt(inimg, outimg2);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&runTime, start, stop);
    cout << "A3() time is " << (runTime) / 100 << " ms" << endl;
    ImageBasicOp::copyToHost(outimg2);
    ImageBasicOp::writeToFile("A3_outimg.bmp", outimg2); 

    // // Pattern 表法，Pattern位于常量内存
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);
    // for (int i = 0; i < 100; i++) 
    //     thin_gpu.thinHanPtCon(inimg, outimg3);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&runTime, start, stop);
    // cout << "A4() time is " << (runTime) / 100 << " ms" << endl;
    // ImageBasicOp::copyToHost(outimg3);
    // ImageBasicOp::writeToFile("A4_outimg.bmp", outimg3); 

    // // 直接并行,一个线程处理四个点
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);
    // for (int i = 0; i < 100; i++) 
    //     thin_gpu.thinHanFour(inimg, outimg4);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&runTime, start, stop);
    // cout << "A5() time is " << (runTime) / 100 << " ms" << endl;
    // ImageBasicOp::copyToHost(outimg4);
    // ImageBasicOp::writeToFile("A5_outimg.bmp", outimg4); 

    // // Pattern 表法，Pattern位于常量内存,一个线程处理四个点
    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // hipEventRecord(start, 0);
    // for (int i = 0; i < 100; i++) 
    //     thin_gpu.thinHanPtConFour(inimg, outimg5);
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&runTime, start, stop);
    // cout << "A5() time is " << (runTime) / 100 << " ms" << endl;
    // ImageBasicOp::copyToHost(outimg5);
    // ImageBasicOp::writeToFile("A6_outimg.bmp", outimg5); 
    
    ImageBasicOp::deleteImage(inimg);
    ImageBasicOp::deleteImage(outimg1);
    ImageBasicOp::deleteImage(outimg2);
    ImageBasicOp::deleteImage(outimg3);
    ImageBasicOp::deleteImage(outimg4);
    ImageBasicOp::deleteImage(outimg5);
    ImageBasicOp::deleteImage(outimg6);

	return 0;
}
