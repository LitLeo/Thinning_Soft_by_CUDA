#include "hip/hip_runtime.h"
#include "Thinning.h"
#include <iostream>
using namespace std;

static __global__ void _thinGpuIter1Ker(ImageCuda outimg, ImageCuda tempimg, int *devchangecount,
                                            unsigned char highpixel, unsigned char lowpixel)
{
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = blockIdx.y * blockDim.y + threadIdx.y;

    if (dstc >= tempimg.imgMeta.width - 1 || 
        dstr >= tempimg.imgMeta.height - 1 || dstc < 1 || dstr < 1)
        return;

    unsigned char *outptr;
    int curpos = dstr * tempimg.pitchBytes + dstc;
    outptr = tempimg.imgMeta.imgData + curpos;

    if (*outptr != lowpixel) {
        int row1 = (dstr - 1) * tempimg.pitchBytes;
        int row2 = row1 + tempimg.pitchBytes;
        int row3 = row2 + tempimg.pitchBytes;

        unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == highpixel;
        unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == highpixel;
        unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == highpixel;
        unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == highpixel;
        unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == highpixel;
        unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == highpixel;
        unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == highpixel;
        unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == highpixel;

        int C  = (!p2 & (p3 | p4)) + (!p4 & (p5 | p6)) +
                 (!p6 & (p7 | p8)) + (!p8 & (p1 | p2));
        int N1 = (p1 | p2) + (p3 | p4) + (p5 | p6) + (p7 | p8);
        int N2 = (p2 | p3) + (p4 | p5) + (p6 | p7) + (p8 | p1);
        int N  = N1 < N2 ? N1 : N2;
        int m  = ((p6 | p7 | !p1) & p8);

        if (C == 1 && (N >= 2 && N <= 3) && m == 0) {
            outimg.imgMeta.imgData[curpos] = lowpixel;
            *devchangecount = 1;
        } 
    }
}

static __global__ void _thinGpuIter2Ker(ImageCuda tempimg, ImageCuda outimg, 
                                    int *devchangecount, unsigned char highpixel,
                                    unsigned char lowpixel)
{
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = blockIdx.y * blockDim.y + threadIdx.y;

    if (dstc >= tempimg.imgMeta.width - 1 || 
        dstr >= tempimg.imgMeta.height - 1 || dstc < 1 || dstr < 1)
        return;

    unsigned char *outptr;
    int curpos = dstr * tempimg.pitchBytes + dstc;
    outptr = tempimg.imgMeta.imgData + curpos;

    if (*outptr != lowpixel) {
        int row1 = (dstr - 1) * tempimg.pitchBytes;
        int row2 = row1 + tempimg.pitchBytes;
        int row3 = row2 + tempimg.pitchBytes;

        unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == highpixel;
        unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == highpixel;
        unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == highpixel;
        unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == highpixel;
        unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == highpixel;
        unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == highpixel;
        unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == highpixel;
        unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == highpixel;

        int C  = (!p2 & (p3 | p4)) + (!p4 & (p5 | p6)) +
              (!p6 & (p7 | p8)) + (!p8 & (p1 | p2));
        int N1 = (p1 | p2) + (p3 | p4) + (p5 | p6) + (p7 | p8);
        int N2 = (p2 | p3) + (p4 | p5) + (p6 | p7) + (p8 | p1);
        int N  = N1 < N2 ? N1 : N2;
        int m  = ((p2 | p3 | !p5) & p4);

        if (C == 1 && (N >= 2 && N <= 3) && m == 0) {
            outimg.imgMeta.imgData[curpos] = lowpixel;
            *devchangecount = 1;
        }
    }
}

__host__ int Thinning::thinGpu(Image *inimg, Image *outimg)
{
    int errcode;  
    hipError_t cudaerrcode; 

    if (inimg == NULL || outimg == NULL)
     return NULL_POINTER;

    int *devchangecount = NULL;
    int changeCount;
    cudaerrcode = hipMalloc((void **)&devchangecount, sizeof (int));
    if (cudaerrcode != hipSuccess) {
        return CUDA_ERROR;
    }

    Image *tempimg = NULL;
    errcode = ImageBasicOp::newImage(&tempimg);
    if (errcode != NO_ERROR)
        return errcode;
    errcode = ImageBasicOp::makeAtCurrentDevice(tempimg, inimg->width, 
                                             inimg->height);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    ImageCuda tempsubimgCud;
    errcode = ImageBasicOp::roiSubImage(tempimg, &tempsubimgCud);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    dim3 gridsize, blocksize;
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y - 1) / blocksize.y;

    changeCount = 1;
    // int iter_num = 0;
    while (changeCount > 0) {
        // iter_num ++;
        changeCount = 0;
        cudaerrcode = hipMemcpy(devchangecount, &changeCount, sizeof (int),
                              hipMemcpyHostToDevice);
        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }

        cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
                                  outimg->imgData, outsubimgCud.deviceId, 
                                  outsubimgCud.pitchBytes * outimg->height);

        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }

        _thinGpuIter1Ker<<<gridsize, blocksize>>>(outsubimgCud, tempsubimgCud, devchangecount,
                                                     highPixel, lowPixel);
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }

        cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
                                  outimg->imgData, outsubimgCud.deviceId, 
                                  outsubimgCud.pitchBytes * outimg->height);

        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }

        _thinGpuIter2Ker<<<gridsize, blocksize>>>(tempsubimgCud, outsubimgCud,
                                                     devchangecount, highPixel, lowPixel);
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }     

        cudaerrcode = hipMemcpy(&changeCount, devchangecount, sizeof (int),
                              hipMemcpyDeviceToHost);
        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }

    }
    // cout << "thinGH iter_num = " << iter_num << endl;
    hipFree(devchangecount);
    ImageBasicOp::deleteImage(tempimg);

    return NO_ERROR;

}

static __global__ void _thinGpuFourIter1Ker(ImageCuda outimg, ImageCuda tempimg, int *devchangecount,
                                            unsigned char highpixel, unsigned char lowpixel)
{
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = (blockIdx.y * blockDim.y + threadIdx.y) * 4;

    if (dstc >= tempimg.imgMeta.width - 1 || 
        dstr >= tempimg.imgMeta.height - 1 || dstc < 1 || dstr < 1)
        return;

    unsigned char *outptr;

    int curpos = dstr * tempimg.pitchBytes + dstc;

    outptr = tempimg.imgMeta.imgData + curpos;

    if (*outptr != lowpixel) {
        int row1 = (dstr - 1) * tempimg.pitchBytes;
        int row2 = row1 + tempimg.pitchBytes;
        int row3 = row2 + tempimg.pitchBytes;

        unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == highpixel;
        unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == highpixel;
        unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == highpixel;
        unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == highpixel;
        unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == highpixel;
        unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == highpixel;
        unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == highpixel;
        unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == highpixel;

        int C  = (!p2 & (p3 | p4)) + (!p4 & (p5 | p6)) +
                 (!p6 & (p7 | p8)) + (!p8 & (p1 | p2));
        int N1 = (p1 | p2) + (p3 | p4) + (p5 | p6) + (p7 | p8);
        int N2 = (p2 | p3) + (p4 | p5) + (p6 | p7) + (p8 | p1);
        int N  = N1 < N2 ? N1 : N2;
        int m  = ((p6 | p7 | !p1) & p8);

        if (C == 1 && (N >= 2 && N <= 3) && m == 0) {
            outimg.imgMeta.imgData[curpos] = lowpixel;
            *devchangecount = 1;
        } 
    }

    for (int i = 0; i < 3; ++i) {
        if (++dstr > tempimg.imgMeta.height - 1)
        return ;
        curpos += tempimg.pitchBytes;  

        outptr = tempimg.imgMeta.imgData + curpos;

        if (*outptr != lowpixel) {
            int row1 = (dstr - 1) * tempimg.pitchBytes;
            int row2 = row1 + tempimg.pitchBytes;
            int row3 = row2 + tempimg.pitchBytes;

            unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == highpixel;
            unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == highpixel;
            unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == highpixel;
            unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == highpixel;
            unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == highpixel;
            unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == highpixel;
            unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == highpixel;
            unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == highpixel;


            int C  = (!p2 & (p3 | p4)) + (!p4 & (p5 | p6)) + (!p6 & (p7 | p8)) + (!p8 & (p1 | p2));
            int N1 = (p1 | p2) + (p3 | p4) + (p5 | p6) + (p7 | p8);
            int N2 = (p2 | p3) + (p4 | p5) + (p6 | p7) + (p8 | p1);
            int N  = N1 < N2 ? N1 : N2;
            int m  = ((p6 | p7 | !p1) & p8);

            if (C == 1 && (N >= 2 && N <= 3) && m == 0) {
                outimg.imgMeta.imgData[curpos] = lowpixel;
                *devchangecount = 1;
            }
        }
    }
}

static __global__ void _thinGpuFourIter2Ker(ImageCuda tempimg, ImageCuda outimg, 
        int *devchangecount, unsigned char highpixel, unsigned char lowpixel)
{
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = (blockIdx.y * blockDim.y + threadIdx.y) * 4;

    if (dstc >= tempimg.imgMeta.width - 1 || 
        dstr >= tempimg.imgMeta.height - 1 || dstc < 1 || dstr < 1)
        return;

    unsigned char *outptr;

    int curpos = dstr * tempimg.pitchBytes + dstc;

    outptr = tempimg.imgMeta.imgData + curpos;

    if (*outptr != lowpixel) {
        int row1 = (dstr - 1) * tempimg.pitchBytes;
        int row2 = row1 + tempimg.pitchBytes;
        int row3 = row2 + tempimg.pitchBytes;

        unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == highpixel;
        unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == highpixel;
        unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == highpixel;
        unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == highpixel;
        unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == highpixel;
        unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == highpixel;
        unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == highpixel;
        unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == highpixel;


        int C  = (!p2 & (p3 | p4)) + (!p4 & (p5 | p6)) +
        (!p6 & (p7 | p8)) + (!p8 & (p1 | p2));
        int N1 = (p1 | p2) + (p3 | p4) + (p5 | p6) + (p7 | p8);
        int N2 = (p2 | p3) + (p4 | p5) + (p6 | p7) + (p8 | p1);
        int N  = N1 < N2 ? N1 : N2;
        int m  = ((p2 | p3 | !p5) & p4);

        if (C == 1 && (N >= 2 && N <= 3) && m == 0) {
            outimg.imgMeta.imgData[curpos] = lowpixel;
            *devchangecount = 1;
        }
    }

    for (int i = 0; i < 3; ++i) {
        if (++dstr > tempimg.imgMeta.height - 1)
            return ;
        curpos += tempimg.pitchBytes;  

        outptr = tempimg.imgMeta.imgData + curpos;

        if (*outptr != lowpixel) {
            int row1 = (dstr - 1) * tempimg.pitchBytes;
            int row2 = row1 + tempimg.pitchBytes;
            int row3 = row2 + tempimg.pitchBytes;

            unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == highpixel;
            unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == highpixel;
            unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == highpixel;
            unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == highpixel;
            unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == highpixel;
            unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == highpixel;
            unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == highpixel;
            unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == highpixel;

            int C  = (!p2 & (p3 | p4)) + (!p4 & (p5 | p6)) +
            (!p6 & (p7 | p8)) + (!p8 & (p1 | p2));
            int N1 = (p1 | p2) + (p3 | p4) + (p5 | p6) + (p7 | p8);
            int N2 = (p2 | p3) + (p4 | p5) + (p6 | p7) + (p8 | p1);
            int N  = N1 < N2 ? N1 : N2;
            int m  = ((p2 | p3 | !p5) & p4);

            if (C == 1 && (N >= 2 && N <= 3) && m == 0) {
                outimg.imgMeta.imgData[curpos] = lowpixel;
                *devchangecount = 1;
            }
        }
    }
}

__host__ int Thinning::thinGpuFour(Image *inimg, Image *outimg)
{
    int errcode;  
    hipError_t cudaerrcode; 

    if (inimg == NULL || outimg == NULL)
    return NULL_POINTER;

    Image *tempimg = NULL;
    int *devchangecount = NULL;

    int changeCount;

    cudaerrcode = hipMalloc((void **)&devchangecount, sizeof (int));
    if (cudaerrcode != hipSuccess) 
        return CUDA_ERROR;

    errcode = ImageBasicOp::newImage(&tempimg);
    if (errcode != NO_ERROR)
        return errcode;
    errcode = ImageBasicOp::makeAtCurrentDevice(tempimg, inimg->width, inimg->height);
    if (errcode != NO_ERROR) 
        return errcode;

    errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
    if (errcode != NO_ERROR) 
        return errcode;

    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR) 
        return errcode;

    ImageCuda tempsubimgCud;
    errcode = ImageBasicOp::roiSubImage(tempimg, &tempsubimgCud);
    if (errcode != NO_ERROR) 
        return errcode;

    dim3 gridsize, blocksize;
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y * 4 - 1) / blocksize.y * 4;

    changeCount = 1;

    while (changeCount > 0) {
        changeCount = 0;
        cudaerrcode = hipMemcpy(devchangecount, &changeCount, sizeof (int),
        hipMemcpyHostToDevice);
        if (cudaerrcode != hipSuccess) 
            return CUDA_ERROR;

        cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
        outimg->imgData, outsubimgCud.deviceId, 
        outsubimgCud.pitchBytes * outimg->height);

        if (cudaerrcode != hipSuccess) 
            return CUDA_ERROR;

        _thinGpuFourIter1Ker<<<gridsize, blocksize>>>(outsubimgCud, tempsubimgCud, devchangecount,
        highPixel, lowPixel);
        if (hipGetLastError() != hipSuccess) 
            return CUDA_ERROR;

        cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
        outimg->imgData, outsubimgCud.deviceId, 
        outsubimgCud.pitchBytes * outimg->height);

        if (cudaerrcode != hipSuccess) 
            return CUDA_ERROR;

        _thinGpuFourIter2Ker<<<gridsize, blocksize>>>(tempsubimgCud, outsubimgCud,
        devchangecount, highPixel, lowPixel);
        if (hipGetLastError() != hipSuccess) 
            return CUDA_ERROR;

        cudaerrcode = hipMemcpy(&changeCount, devchangecount, sizeof (int),
        hipMemcpyDeviceToHost);
        if (cudaerrcode != hipSuccess) 
            return CUDA_ERROR;

    }

    hipFree(devchangecount);
    ImageBasicOp::deleteImage(tempimg);

    return NO_ERROR;

}

static __global__ void _thinGpuPtIter1Ker(ImageCuda outimg,
                                         ImageCuda tempimg,
                                         unsigned char *devlutthin,
                                         int *devchangecount,
                                         unsigned char highpixel, 
                                         unsigned char lowpixel)
{
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = blockIdx.y * blockDim.y + threadIdx.y;

    if (dstc >= tempimg.imgMeta.width - 1 || 
        dstr >= tempimg.imgMeta.height - 1 || dstc < 1 || dstr < 1)
        return;

    unsigned char *outptr;

    int curpos = dstr * tempimg.pitchBytes + dstc;

    outptr = tempimg.imgMeta.imgData + curpos ;

    if (*outptr != lowpixel) {
        int index = 0;

        int row1 = (dstr - 1) * tempimg.pitchBytes;
        int row2 = row1 + tempimg.pitchBytes;
        int row3 = row2 + tempimg.pitchBytes;

        unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == highpixel;
        unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == highpixel;
        unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == highpixel;
        unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == highpixel;
        unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == highpixel;
        unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == highpixel;
        unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == highpixel;
        unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == highpixel;

        index = p1 * 1 + p2 * 2 + p3 * 4 + p4 * 8 + p5 * 16 + p6 * 32 + p7 * 64 + p8 * 128;
        
        if (devlutthin[index]) {
            outimg.imgMeta.imgData[curpos] = lowpixel;
            *devchangecount = 1;
        }
    }
}

static __global__ void _thinGpuPtIter2Ker(ImageCuda tempimg,
                                         ImageCuda outimg,
                                         unsigned char *devlutthin,
                                         int *devchangecount,
                                         unsigned char lowpixel)
{
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = blockIdx.y * blockDim.y + threadIdx.y;

    if (dstc >= tempimg.imgMeta.width - 1 || 
        dstr >= tempimg.imgMeta.height - 1 || dstc < 1 || dstr < 1)
        return;

    unsigned char *temptr;

    int curpos = dstr * outimg.pitchBytes + dstc;
    
    temptr = tempimg.imgMeta.imgData + curpos;


    if (*temptr != lowpixel) {
        int index = 0;

        int row1 = (dstr - 1) * tempimg.pitchBytes;
        int row2 = row1 + tempimg.pitchBytes;
        int row3 = row2 + tempimg.pitchBytes;

        unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == 255;
        unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == 255;
        unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == 255;
        unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == 255;
        unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == 255;
        unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == 255;
        unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == 255;
        unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == 255;

        index = p1 * 1 + p2 * 2 + p3 * 4 + p4 * 8 + p5 * 16 + p6 * 32 + p7 * 64 + p8 * 128;

        if (devlutthin[index + 256]) {
            outimg.imgMeta.imgData[curpos] = lowpixel;
            *devchangecount = 1;
        }
    }    
}

__host__ int Thinning::thinGpuPt (Image *inimg, Image *outimg)
{
    int errcode;  
    hipError_t cudaerrcode; 

    if (inimg == NULL || outimg == NULL)
        return NULL_POINTER;

    unsigned char *devlutthin = NULL;
    Image *tempimg = NULL;
    int *devchangecount = NULL;

    cudaerrcode = hipMalloc((void **)&devlutthin, 
                             512 *  sizeof (unsigned char));
    if (cudaerrcode != hipSuccess) 
        return CUDA_ERROR;

    unsigned char lutthin[] = { 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 1, 0, 1, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
                                0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0};

    cudaerrcode = hipMemcpy(devlutthin, lutthin, 512 * sizeof (unsigned char), 
                             hipMemcpyHostToDevice);
    if (cudaerrcode != hipSuccess) {
        return CUDA_ERROR;
    }

    int changeCount;

    cudaerrcode = hipMalloc((void **)&devchangecount, sizeof (int));
    if (cudaerrcode != hipSuccess) {
        return CUDA_ERROR;
    }

    errcode = ImageBasicOp::newImage(&tempimg);
    if (errcode != NO_ERROR)
        return errcode;
    errcode = ImageBasicOp::makeAtCurrentDevice(tempimg, inimg->width, 
                                                inimg->height);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    ImageCuda tempsubimgCud;
    errcode = ImageBasicOp::roiSubImage(tempimg, &tempsubimgCud);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    dim3 gridsize, blocksize;
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y - 1) / blocksize.y;

    changeCount = 1;

    while (changeCount > 0) {
        changeCount = 0;
        cudaerrcode = hipMemcpy(devchangecount, &changeCount, sizeof (int),
                                 hipMemcpyHostToDevice);
        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }
        cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
                                     outimg->imgData, outsubimgCud.deviceId, 
                                     outsubimgCud.pitchBytes * outimg->height);
       
        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }
        
        _thinGpuPtIter1Ker<<<gridsize, blocksize>>>(outsubimgCud, tempsubimgCud, devlutthin, devchangecount, highPixel, lowPixel);
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }

        cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
                                     outimg->imgData, outsubimgCud.deviceId, 
                                     outsubimgCud.pitchBytes * outimg->height);
       
        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }
        _thinGpuPtIter2Ker<<<gridsize, blocksize>>>(tempsubimgCud, outsubimgCud,
                                                   devlutthin, devchangecount,
                                                   lowPixel);
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }     
        
        cudaerrcode = hipMemcpy(&changeCount, devchangecount, sizeof (int),
                                 hipMemcpyDeviceToHost);
        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }

   }

    hipFree(devlutthin);
    hipFree(devchangecount);
    ImageBasicOp::deleteImage(tempimg);

    return NO_ERROR;
}

static __global__ void _thinGpuPtFourIter1Ker(ImageCuda outimg,
                                         ImageCuda tempimg,
                                         int *devchangecount,
                                         unsigned char *devlutthin,
                                         unsigned char highpixel, 
                                         unsigned char lowpixel)
{
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = (blockIdx.y * blockDim.y + threadIdx.y) * 4;

    if (dstc >= tempimg.imgMeta.width - 1 || 
        dstr >= tempimg.imgMeta.height - 1 || dstc < 1 || dstr < 1)
        return;

    unsigned char *outptr;

    int curpos = dstr * tempimg.pitchBytes + dstc;

    outptr = tempimg.imgMeta.imgData + curpos ;

    if (*outptr != lowpixel) {
        int index = 0;

        int row1 = (dstr - 1) * tempimg.pitchBytes;
        int row2 = row1 + tempimg.pitchBytes;
        int row3 = row2 + tempimg.pitchBytes;

        unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == 255;
        unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == 255;
        unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == 255;
        unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == 255;
        unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == 255;
        unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == 255;
        unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == 255;
        unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == 255;

        index = p1 * 1 + p2 * 2 + p3 * 4 + p4 * 8 + p5 * 16 + p6 * 32 + p7 * 64 + p8 * 128;
        
        if (devlutthin[index]) {
            outimg.imgMeta.imgData[curpos] = lowpixel;
            *devchangecount = 1;
        }
    }

    for (int i = 0; i < 3; ++i) {
        if (++dstr >= tempimg.imgMeta.height - 1) 
            return ;

        curpos += tempimg.pitchBytes;

        outptr = tempimg.imgMeta.imgData + curpos ;

        if (*outptr != lowpixel) {
            int index = 0;

            int row1 = (dstr - 1) * tempimg.pitchBytes;
            int row2 = row1 + tempimg.pitchBytes;
            int row3 = row2 + tempimg.pitchBytes;

            unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == 255;
            unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == 255;
            unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == 255;
            unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == 255;
            unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == 255;
            unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == 255;
            unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == 255;
            unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == 255;

        index = p1 * 1 + p2 * 2 + p3 * 4 + p4 * 8 + p5 * 16 + p6 * 32 + p7 * 64 + p8 * 128;
            
            if (devlutthin[index]) {
                outimg.imgMeta.imgData[curpos] = lowpixel;
                *devchangecount = 1;
            }
        }
    }
}

static __global__ void _thinGpuPtFourIter2Ker(ImageCuda tempimg,
                                         ImageCuda outimg,
                                         unsigned char *devlutthin,
                                         int *devchangecount,
                                         unsigned char lowpixel)
{
    int dstc = blockIdx.x * blockDim.x + threadIdx.x;
    int dstr = (blockIdx.y * blockDim.y + threadIdx.y) * 4;

    if (dstc >= tempimg.imgMeta.width - 1 || 
        dstr >= tempimg.imgMeta.height - 1 || dstc < 1 || dstr < 1)
        return;

    unsigned char *outptr;

    int curpos = dstr * outimg.pitchBytes + dstc;
    
    outptr = tempimg.imgMeta.imgData + curpos;


    if (*outptr != lowpixel) {
        int index = 0;

        int row1 = (dstr - 1) * tempimg.pitchBytes;
        int row2 = row1 + tempimg.pitchBytes;
        int row3 = row2 + tempimg.pitchBytes;

        unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == 255;
        unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == 255;
        unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == 255;
        unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == 255;
        unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == 255;
        unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == 255;
        unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == 255;
        unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == 255;

        index = p1 * 1 + p2 * 2 + p3 * 4 + p4 * 8 + p5 * 16 + p6 * 32 + p7 * 64 + p8 * 128;

        if (devlutthin[index + 256]) {
            outimg.imgMeta.imgData[curpos] = lowpixel;
            *devchangecount = 1;
        }
    }  

    for (int i = 0; i < 3; ++i) {
        if (++dstr >= tempimg.imgMeta.height - 1) 
            return ;

        curpos += tempimg.pitchBytes;

        outptr = tempimg.imgMeta.imgData + curpos ;

         if (*outptr != lowpixel) {
            int index = 0;

            int row1 = (dstr - 1) * tempimg.pitchBytes;
            int row2 = row1 + tempimg.pitchBytes;
            int row3 = row2 + tempimg.pitchBytes;

            unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == 255;
            unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == 255;
            unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == 255;
            unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == 255;
            unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == 255;
            unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == 255;
            unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == 255;
            unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == 255;

        index = p1 * 1 + p2 * 2 + p3 * 4 + p4 * 8 + p5 * 16 + p6 * 32 + p7 * 64 + p8 * 128;

            if (devlutthin[index + 256]) {
                outimg.imgMeta.imgData[curpos] = lowpixel;
                *devchangecount = 1;
            }
        } 
    }  
}

__host__ int Thinning::thinGpuPtFour (Image *inimg, Image *outimg)
{
    int errcode;  
    hipError_t cudaerrcode; 

    if (inimg == NULL || outimg == NULL)
        return NULL_POINTER;

    unsigned char *devlutthin = NULL;
    Image *tempimg = NULL;
    int *devchangecount = NULL;

    cudaerrcode = hipMalloc((void **)&devlutthin, 
                             512 *  sizeof (unsigned char));
    if (cudaerrcode != hipSuccess) 
        return CUDA_ERROR;

    unsigned char lutthin[] = { 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 1, 0, 1, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
                                0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0};

    cudaerrcode = hipMemcpy(devlutthin, lutthin, 512 * sizeof (unsigned char), 
                             hipMemcpyHostToDevice);
    if (cudaerrcode != hipSuccess) {
        return CUDA_ERROR;
    }

    int changeCount;

    cudaerrcode = hipMalloc((void **)&devchangecount, sizeof (int));
    if (cudaerrcode != hipSuccess) {
        return CUDA_ERROR;
    }

    errcode = ImageBasicOp::newImage(&tempimg);
    if (errcode != NO_ERROR)
        return errcode;
    errcode = ImageBasicOp::makeAtCurrentDevice(tempimg, inimg->width, 
                                                inimg->height);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    ImageCuda outsubimgCud;
    errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    ImageCuda tempsubimgCud;
    errcode = ImageBasicOp::roiSubImage(tempimg, &tempsubimgCud);
    if (errcode != NO_ERROR) {
        return errcode;
    }

    dim3 gridsize, blocksize;
    blocksize.x = DEF_BLOCK_X;
    blocksize.y = DEF_BLOCK_Y;
    gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
    gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y * 4 - 1) / blocksize.y * 4;

    changeCount = 1;

    while (changeCount > 0) {
        changeCount = 0;
        cudaerrcode = hipMemcpy(devchangecount, &changeCount, sizeof (int),
                                 hipMemcpyHostToDevice);
        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }
        cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
                                     outimg->imgData, outsubimgCud.deviceId, 
                                     outsubimgCud.pitchBytes * outimg->height);
       
        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }
        
        _thinGpuPtFourIter1Ker<<<gridsize, blocksize>>>(outsubimgCud, tempsubimgCud, devchangecount, 
                                                   devlutthin, highPixel,
                                                   lowPixel);
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }

        cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
                                     outimg->imgData, outsubimgCud.deviceId, 
                                     outsubimgCud.pitchBytes * outimg->height);
       
        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }
        _thinGpuPtFourIter2Ker<<<gridsize, blocksize>>>(tempsubimgCud, outsubimgCud,
                                                   devlutthin, devchangecount,
                                                   lowPixel);
        if (hipGetLastError() != hipSuccess) {
            return CUDA_ERROR;
        }     
        
        cudaerrcode = hipMemcpy(&changeCount, devchangecount, sizeof (int),
                                 hipMemcpyDeviceToHost);
        if (cudaerrcode != hipSuccess) {
            return CUDA_ERROR;
        }

   }

    hipFree(devlutthin);
    hipFree(devchangecount);
    ImageBasicOp::deleteImage(tempimg);

    return NO_ERROR;
}

// __constant__ unsigned char devlutthin[] = { 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 1, 0, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 0, 1, 0, 1, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 
//                                 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 0, 0, 0, 0, 0, 0};


// static __global__ void _thinGpuPtFourIter1Ker(ImageCuda outimg,
//                                          ImageCuda tempimg,
//                                          int *devchangecount,
//                                          unsigned char highpixel, 
//                                          unsigned char lowpixel)
// {
//     int dstc = blockIdx.x * blockDim.x + threadIdx.x;
//     int dstr = blockIdx.y * blockDim.y + threadIdx.y;

//     if (dstc >= tempimg.imgMeta.width - 1 || 
//         dstr >= tempimg.imgMeta.height - 1 || dstc < 1 || dstr < 1)
//         return;

//     unsigned char *outptr;

//     int curpos = dstr * tempimg.pitchBytes + dstc;

//     outptr = tempimg.imgMeta.imgData + curpos ;

//     if (*outptr != lowpixel) {
//         int index = 0;

//         int row1 = (dstr - 1) * tempimg.pitchBytes;
//         int row2 = row1 + tempimg.pitchBytes;
//         int row3 = row2 + tempimg.pitchBytes;

//         unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == 255;
//         unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == 255;
//         unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == 255;
//         unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == 255;
//         unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == 255;
//         unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == 255;
//         unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == 255;
//         unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == 255;

//         index = p1 * 1 + p2 * 2 + p3 * 4 + p4 * 8 + p5 * 16 + p6 * 32 + p7 * 64 + p8 * 128;
        
//         if (devlutthin[index]) {
//             outimg.imgMeta.imgData[curpos] = lowpixel;
//             *devchangecount = 1;
//         }
//     }

//     for (int i = 0; i < 3; ++i) {
//         if (++dstr >= tempimg.imgMeta.height - 1) 
//             return ;

//         curpos += tempimg.pitchBytes;

//         outptr = tempimg.imgMeta.imgData + curpos ;

//         if (*outptr != lowpixel) {
//             int index = 0;

//             int row1 = (dstr - 1) * tempimg.pitchBytes;
//             int row2 = row1 + tempimg.pitchBytes;
//             int row3 = row2 + tempimg.pitchBytes;

//             unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == 255;
//             unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == 255;
//             unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == 255;
//             unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == 255;
//             unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == 255;
//             unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == 255;
//             unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == 255;
//             unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == 255;

//         index = p1 * 1 + p2 * 2 + p3 * 4 + p4 * 8 + p5 * 16 + p6 * 32 + p7 * 64 + p8 * 128;
            
//             if (devlutthin[index]) {
//                 outimg.imgMeta.imgData[curpos] = lowpixel;
//                 *devchangecount = 1;
//             }
//         }
//     }
// }

// static __global__ void _thinGpuPtFourIter2Ker(ImageCuda tempimg,
//                                          ImageCuda outimg,
//                                          int *devchangecount,
//                                          unsigned char lowpixel)
// {
//     int dstc = blockIdx.x * blockDim.x + threadIdx.x;
//     int dstr = blockIdx.y * blockDim.y + threadIdx.y;

//     if (dstc >= tempimg.imgMeta.width - 1 || 
//         dstr >= tempimg.imgMeta.height - 1 || dstc < 1 || dstr < 1)
//         return;

//     unsigned char *outptr;

//     int curpos = dstr * outimg.pitchBytes + dstc;
    
//     outptr = tempimg.imgMeta.imgData + curpos;


//     if (*outptr != lowpixel) {
//         int index = 0;

//         int row1 = (dstr - 1) * tempimg.pitchBytes;
//         int row2 = row1 + tempimg.pitchBytes;
//         int row3 = row2 + tempimg.pitchBytes;

//         unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == 255;
//         unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == 255;
//         unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == 255;
//         unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == 255;
//         unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == 255;
//         unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == 255;
//         unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == 255;
//         unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == 255;

//         index = p1 * 1 + p2 * 2 + p3 * 4 + p4 * 8 + p5 * 16 + p6 * 32 + p7 * 64 + p8 * 128;

//         if (devlutthin[index + 256]) {
//             outimg.imgMeta.imgData[curpos] = lowpixel;
//             *devchangecount = 1;
//         }
//     }  

//     for (int i = 0; i < 3; ++i) {
//         if (++dstr >= tempimg.imgMeta.height - 1) 
//             return ;

//         curpos += tempimg.pitchBytes;

//         outptr = tempimg.imgMeta.imgData + curpos ;

//          if (*outptr != lowpixel) {
//             int index = 0;

//             int row1 = (dstr - 1) * tempimg.pitchBytes;
//             int row2 = row1 + tempimg.pitchBytes;
//             int row3 = row2 + tempimg.pitchBytes;

//             unsigned char p1 = tempimg.imgMeta.imgData[dstc-1 + row1] == 255;
//             unsigned char p2 = tempimg.imgMeta.imgData[dstc+    row1] == 255;
//             unsigned char p3 = tempimg.imgMeta.imgData[dstc+1 + row1] == 255;
//             unsigned char p4 = tempimg.imgMeta.imgData[dstc+1 + row2] == 255;
//             unsigned char p5 = tempimg.imgMeta.imgData[dstc+1 + row3] == 255;
//             unsigned char p6 = tempimg.imgMeta.imgData[dstc+    row3] == 255;
//             unsigned char p7 = tempimg.imgMeta.imgData[dstc-1 + row3] == 255;
//             unsigned char p8 = tempimg.imgMeta.imgData[dstc-1 + row2] == 255;

//         index = p1 * 1 + p2 * 2 + p3 * 4 + p4 * 8 + p5 * 16 + p6 * 32 + p7 * 64 + p8 * 128;

//             if (devlutthin[index + 256]) {
//                 outimg.imgMeta.imgData[curpos] = lowpixel;
//                 *devchangecount = 1;
//             }
//         } 
//     }  
// }

// __host__ int Thinning::thinGpuPtFour (Image *inimg, Image *outimg)
// {
//     int errcode;  
//     hipError_t cudaerrcode; 

//     if (inimg == NULL || outimg == NULL)
//         return NULL_POINTER;

//     unsigned char *devlutthin = NULL;
//     Image *tempimg = NULL;
//     int *devchangecount = NULL;

//     int changeCount;

//     cudaerrcode = hipMalloc((void **)&devchangecount, sizeof (int));
//     if (cudaerrcode != hipSuccess) {
//         return CUDA_ERROR;
//     }

//     errcode = ImageBasicOp::newImage(&tempimg);
//     if (errcode != NO_ERROR)
//         return errcode;
//     errcode = ImageBasicOp::makeAtCurrentDevice(tempimg, inimg->width, 
//                                                 inimg->height);
//     if (errcode != NO_ERROR) {
//         return errcode;
//     }

//     errcode = ImageBasicOp::copyToCurrentDevice(inimg, outimg);
//     if (errcode != NO_ERROR) {
//         return errcode;
//     }

//     ImageCuda outsubimgCud;
//     errcode = ImageBasicOp::roiSubImage(outimg, &outsubimgCud);
//     if (errcode != NO_ERROR) {
//         return errcode;
//     }

//     ImageCuda tempsubimgCud;
//     errcode = ImageBasicOp::roiSubImage(tempimg, &tempsubimgCud);
//     if (errcode != NO_ERROR) {
//         return errcode;
//     }

//     dim3 gridsize, blocksize;
//     blocksize.x = DEF_BLOCK_X;
//     blocksize.y = DEF_BLOCK_Y;
//     gridsize.x = (outsubimgCud.imgMeta.width + blocksize.x - 1) / blocksize.x;
//     gridsize.y = (outsubimgCud.imgMeta.height + blocksize.y * 4 - 1) / blocksize.y * 4;

//     changeCount = 1;

//     while (changeCount > 0) {
//         changeCount = 0;
//         cudaerrcode = hipMemcpy(devchangecount, &changeCount, sizeof (int),
//                                  hipMemcpyHostToDevice);
//         if (cudaerrcode != hipSuccess) {
//             return CUDA_ERROR;
//         }
//         cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
//                                      outimg->imgData, outsubimgCud.deviceId, 
//                                      outsubimgCud.pitchBytes * outimg->height);
       
//         if (cudaerrcode != hipSuccess) {
//             return CUDA_ERROR;
//         }
        
//         _thinGpuPtFourIter1Ker<<<gridsize, blocksize>>>(outsubimgCud, tempsubimgCud, devchangecount, 
//                                                     highPixel,
//                                                    lowPixel);
//         if (hipGetLastError() != hipSuccess) {
//             return CUDA_ERROR;
//         }

//         cudaerrcode = hipMemcpyPeer(tempimg->imgData, tempsubimgCud.deviceId, 
//                                      outimg->imgData, outsubimgCud.deviceId, 
//                                      outsubimgCud.pitchBytes * outimg->height);
       
//         if (cudaerrcode != hipSuccess) {
//             return CUDA_ERROR;
//         }
//         _thinGpuPtFourIter2Ker<<<gridsize, blocksize>>>(tempsubimgCud, outsubimgCud,
//                                                     devchangecount,
//                                                    lowPixel);
//         if (hipGetLastError() != hipSuccess) {
//             return CUDA_ERROR;
//         }     
        
//         cudaerrcode = hipMemcpy(&changeCount, devchangecount, sizeof (int),
//                                  hipMemcpyDeviceToHost);
//         if (cudaerrcode != hipSuccess) {
//             return CUDA_ERROR;
//         }

//    }

//     hipFree(devlutthin);
//     hipFree(devchangecount);
//     ImageBasicOp::deleteImage(tempimg);

//     return NO_ERROR;
// }