#include "hip/hip_runtime.h"
#include "ErrorCode.h"
#include <stdio.h>

__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
        C[i] = A[i] + B[i];
}

// warmup 函数，用于计时时 warmup GPU，实际是一个 vector 相加
void warmup()
{
    int numElements = 1024;
    size_t size = numElements * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);

    float *d_B = NULL;
    hipMalloc((void **)&d_B, size);

    float *d_C = NULL;
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 32;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

}
